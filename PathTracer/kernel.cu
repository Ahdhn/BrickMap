#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "sunsky.cuh"

#include "assert_cuda.h"
#include "cuda_surface_types.h"
#include ""
#include "surface_functions.h"

#include "cuda_definitions.h"

constexpr int NUM_SPHERES = 7;
constexpr float VERY_FAR = 1e20f;
constexpr int MAX_BOUNCES = 5;

surface<void, cudaSurfaceType2D> surf;
texture<float, hipTextureTypeCubemap> skybox;

//"Xorshift RNGs" by George Marsaglia
//http://excamera.com/sphinx/article-xorshift.html
__device__ unsigned int RandomInt(unsigned int& seed) {
	seed ^= seed << 13;
	seed ^= seed >> 17;
	seed ^= seed << 5;
	return seed;
}

//Random float between [0,1).
__device__ float RandomFloat(unsigned int& seed) {
	return RandomInt(seed) * 2.3283064365387e-10f;
}

__device__ float RandomFloat2(unsigned int& seed) {
	return (RandomInt(seed) >> 16) / 65535.0f;
}

__device__ int RandomIntBetween0AndMax(unsigned int& seed, int max) {
	return int(RandomFloat(seed) * (max + 0.99999f));
}

// A 100% correct but slow implementation
__device__ bool intersect_aabb_correct(const RayQueue& ray, float& tmin) {
	glm::vec3 box_min = { 0, 0, 0 };
	glm::vec3 box_max = { grid_size, grid_size, grid_size };

	tmin = (box_min[0] - ray.origin[0]) / ray.direction.x;
	float tmax = (box_max[0] - ray.origin[0]) / ray.direction.x;

	if (tmin > tmax) {
		float a = tmin;
		tmin = tmax;
		tmax = a;
	}

	float tymin = (box_min[1] - ray.origin[1]) / ray.direction.y;
	float tymax = (box_max[1] - ray.origin[1]) / ray.direction.y;

	if (tymin > tymax) {
		float a = tymin;
		tymin = tymax;
		tymax = a;
	}

	if ((tmin > tymax) || (tymin > tmax))
		return false;

	if (tymin > tmin)
		tmin = tymin;

	if (tymax < tmax)
		tmax = tymax;

	float tzmin = (box_min[2] - ray.origin[2]) / ray.direction.z;
	float tzmax = (box_max[2] - ray.origin[2]) / ray.direction.z;

	if (tzmin > tzmax) {
		float a = tzmin;
		tzmin = tzmax;
		tzmax = a;
	}

	if ((tmin > tzmax) || (tzmin > tmax))
		return false;

	if (tzmin > tmin)
		tmin = tzmin;

	if (tzmax < tmax)
		tmax = tzmax;

	return tmax > glm::max(tmin, 0.f);
}

__device__ inline bool intersect_aabb_branchless(const RayQueue& ray, float& tmin) {
	glm::vec3 box_min = { 0, 0, 0 };
	glm::vec3 box_max = { grid_size, grid_size, grid_size };
	glm::vec3 dir_inv = 1.f / ray.direction;

	float t1 = (box_min[0] - ray.origin[0]) * dir_inv[0];
	float t2 = (box_max[0] - ray.origin[0]) * dir_inv[0];

	tmin = glm::min(t1, t2);
	float tmax = glm::max(t1, t2);

	for (int i = 1; i < 3; ++i) {
		t1 = (box_min[i] - ray.origin[i]) * dir_inv[i];
		t2 = (box_max[i] - ray.origin[i]) * dir_inv[i];

		//tmin = glm::max(tmin, glm::min(t1, t2));
		//tmax = glm::min(tmax, glm::max(t1, t2));
		tmin = glm::max(tmin, glm::min(glm::min(t1, t2), tmax));
		tmax = glm::min(tmax, glm::max(glm::max(t1, t2), tmin));
	}

	return tmax > glm::max(tmin, 0.f);
}

//// From http://www.jcgt.org/published/0006/02/01/
//__device__ bool intersect_aabb_branchless2(const RayQueue& ray, float& tmin) {
//	constexpr glm::vec3 box_min = { 0, 0, 0 };
//	constexpr glm::vec3 box_max = { grid_size, grid_size, grid_size };
//
//	const glm::vec3 t1 = (box_min - ray.origin) / ray.direction;
//	const glm::vec3 t2 = (box_max - ray.origin) / ray.direction;
//	const glm::vec3 tMin = glm::min(t1, t2);
//	const glm::vec3 tMax = glm::max(t1, t2);
//
//	tmin = glm::max(glm::max(tMin.x, 0.f), glm::max(tMin.y, tMin.z));
//	return glm::min(tMax.x, glm::min(tMax.y, tMax.z)) > tmin;
//}

// From http://www.jcgt.org/published/0006/02/01/
//template <typename T>
__device__ bool intersect_aabb_branchless2(const glm::vec3& origin, const glm::vec3& direction, float& tmin) {
	constexpr glm::vec3 box_min = { 0, 0, 0 };
	constexpr glm::vec3 box_max = { grid_size, grid_size, grid_size };

	const glm::vec3 t1 = (box_min - origin) / direction;
	const glm::vec3 t2 = (box_max - origin) / direction;
	const glm::vec3 tMin = glm::min(t1, t2);
	const glm::vec3 tMax = glm::max(t1, t2);

	tmin = glm::max(glm::max(tMin.x, 0.f), glm::max(tMin.y, tMin.z));
	return glm::min(tMax.x, glm::min(tMax.y, tMax.z)) > tmin;
}

//Generate stratified sample of 2D [0,1]^2
__device__ glm::vec2 Random2DStratifiedSample(unsigned int& seed) {
	//Set the size of the pixel in stratums.
	constexpr int width2D = 4;
	constexpr int height2D = 4;
	constexpr float pixelWidth = 1.0f / width2D;
	constexpr float pixelHeight = 1.0f / height2D;

	const int chosenStratum = RandomIntBetween0AndMax(seed, width2D * height2D);
	//Compute stratum X in [0, width-1] and Y in [0,height -1]
	const int stratumX = chosenStratum % width2D;
	const int stratumY = (chosenStratum / width2D) % height2D;

	//Now we split up the pixel into [stratumX,stratumY] pieces.
	//Let's get the width and height of this sample

	const float stratumXStart = pixelWidth * stratumX;
	const float stratumYStart = pixelHeight * stratumY;

	const float randomPointInStratumX = stratumXStart + (RandomFloat(seed) * pixelWidth);
	const float randomPointInStratumY = stratumYStart + (RandomFloat(seed) * pixelHeight);
	return glm::vec2(randomPointInStratumX, randomPointInStratumY);
}

enum Refl_t { DIFF,
			  SPEC,
			  REFR,
			  PHONG,
			  LIGHT };

__device__ inline bool intersect_brick(glm::vec3 origin, const glm::vec3& direction, glm::vec3& normal, float& distance, Brick* brick) {
	origin.x = fmod(origin.x, 8.f);
	origin.y = fmod(origin.y, 8.f);
	origin.z = fmod(origin.z, 8.f);
	//poss = poss % 8;
	// Initialize
	glm::vec3 cb, tmax, tdelta;
	int stepX, outX, X = (int)origin.x;
	int stepY, outY, Y = (int)origin.y;
	int stepZ, outZ, Z = (int)origin.z;


	// Needed because sometimes the AABB intersect returns true while the ray is actually outside slightly. Only happens for faces that touch the AABB sides
	if (X >= cell_size || Y >= cell_size || Z >= cell_size) {
		//printf("mhmm positive\n");
		return false;
	}

	if (X < 0 || Y < 0 || Z < 0) {
		//printf("mhmm negative\n");
		return false;
	}


	if (direction.x > 0) {
		stepX = 1;
		outX = cell_size;
		cb.x = (X + 1);
	} else {
		stepX = -1;
		outX = -1;
		cb.x = X;
	}
	if (direction.y > 0.0f) {
		stepY = 1;
		outY = cell_size;
		cb.y = (Y + 1);
	} else {
		stepY = -1;
		outY = -1;
		cb.y = Y;
	}
	if (direction.z > 0.0f) {
		stepZ = 1;
		outZ = cell_size;
		cb.z = (Z + 1);
	} else {
		stepZ = -1;
		outZ = -1;
		cb.z = Z;
	}
	float rxr, ryr, rzr;
	if (direction.x != 0) {
		rxr = 1.0f / direction.x;
		tmax.x = (cb.x - origin.x) * rxr;
		tdelta.x = stepX * rxr;
	} else
		tmax.x = 1000000;
	if (direction.y != 0) {
		ryr = 1.0f / direction.y;
		tmax.y = (cb.y - origin.y) * ryr;
		tdelta.y = stepY * ryr;
	} else
		tmax.y = 1000000;
	if (direction.z != 0) {
		rzr = 1.0f / direction.z;
		tmax.z = (cb.z - origin.z) * rzr;
		tdelta.z = stepZ * rzr;
	} else
		tmax.z = 1000000;
	distance = 0.f;

	// Stepping through grid
	while (1) {
		int sub_data = (X + Y * cell_size + Z * cell_size * cell_size) / 32;
		int bit = (X + Y * cell_size + Z * cell_size * cell_size) % 32;

		if (sub_data < 0 || sub_data > 15) {
			printf("uwu");
		}

		if (bit < 0 || bit > 31) {
			printf("OwO");
		}

		if (brick->data[sub_data] & (1 << bit)) {
			return true;
		}
	
		if (tmax.x < tmax.y) {
			if (tmax.x < tmax.z) {
				X += stepX;
				if (X == outX)
					return false;
				distance = tmax.x;
				tmax.x += tdelta.x;
				normal = glm::vec3(-stepX, 0, 0);
			} else {
				Z += stepZ;
				if (Z == outZ)
					return false;
				distance = tmax.z;
				tmax.z += tdelta.z;
				normal = glm::vec3(0, 0, -stepZ);
			}
		} else {
			if (tmax.y < tmax.z) {
				Y += stepY;
				if (Y == outY)
					return false;
				distance = tmax.y;
				tmax.y += tdelta.y;
				normal = glm::vec3(0, -stepY, 0);
			} else {
				Z += stepZ;
				if (Z == outZ)
					return false;
				distance = tmax.z;
				tmax.z += tdelta.z;
				normal = glm::vec3(0, 0, -stepZ);
			}
		}
	}
	return false;
}

__device__ inline bool intersect_voxel(glm::vec3 origin, const glm::vec3& direction, glm::vec3& normal, float& distance, Scene::GPUScene scene) {
	// Check if ray hits grid AABB
	float tminn;
	if (!intersect_aabb_branchless2(origin, direction, tminn)) {
		return false;
	}

	// Move ray to hitpoint
	if (tminn > 0) {
		origin += direction * tminn;

		constexpr glm::vec3 grid_center(grid_size / 2.f);
		glm::vec3 to_center = glm::abs(grid_center - origin);
		glm::vec3 signs = glm::sign(origin - grid_center);
		to_center /= glm::max(to_center.x, glm::max(to_center.y, to_center.z));
		normal = signs * glm::trunc(to_center + 0.000001f);

		origin += -normal * epsilon;
	}
	origin /= 8.f;

	// Initialize
	glm::vec3 cb, tmax, tdelta;
	int stepX, outX, X = ((int)origin.x);
	int stepY, outY, Y = ((int)origin.y);
	int stepZ, outZ, Z = ((int)origin.z);

	// Needed because sometimes the AABB intersect returns true while the ray is actually outside slightly. Only happens for faces that touch the AABB sides
	if (X < 0 || X >= cells || Y < 0 || Y >= cells || Z < 0 || Z >= cells) {
		return false;
	}

	if (direction.x > 0) {
		stepX = 1;
		outX = cells;
		cb.x = (X + 1);
	} else {
		stepX = -1;
		outX = -1;
		cb.x = X;
	}
	if (direction.y > 0.0f) {
		stepY = 1;
		outY = cells;
		cb.y = (Y + 1);
	} else {
		stepY = -1;
		outY = -1;
		cb.y = Y;
	}
	if (direction.z > 0.0f) {
		stepZ = 1;
		outZ = cells;
		cb.z = (Z + 1);
	} else {
		stepZ = -1;
		outZ = -1;
		cb.z = Z;
	}
	float rxr, ryr, rzr;
	if (direction.x != 0) {
		rxr = 1.0f / direction.x;
		tmax.x = (cb.x - origin.x) * rxr;
		tdelta.x = stepX * rxr;
	} else
		tmax.x = 1000000;
	if (direction.y != 0) {
		ryr = 1.0f / direction.y;
		tmax.y = (cb.y - origin.y) * ryr;
		tdelta.y = stepY * ryr;
	} else
		tmax.y = 1000000;
	if (direction.z != 0) {
		rzr = 1.0f / direction.z;
		tmax.z = (cb.z - origin.z) * rzr;
		tdelta.z = stepZ * rzr;
	} else
		tmax.z = 1000000;
	distance = 0.f;

	// Stepping through grid
	while (1) {
		Brick* brick = scene.grid[X + Y * cells + Z * cells * cells];
		if (brick != nullptr) {
			float sub_distance = 0.f;
			if (intersect_brick(origin * 8.f + direction * (distance * 8.f + epsilon), direction, normal, sub_distance, brick)) {
				distance += sub_distance + glm::max(tminn, 0.f) + epsilon;
				return true;
			}
			//return true;
		}

		constexpr int step_size = 1;

		if (tmax.x < tmax.y) {
			if (tmax.x < tmax.z) {
				X += stepX;
				if (X == outX)
					return false;
				distance = tmax.x;
				tmax.x += tdelta.x;
				normal = glm::vec3(-stepX, 0, 0);
			} else {
				Z += stepZ;
				if (Z == outZ)
					return false;
				distance = tmax.z;
				tmax.z += tdelta.z;
				normal = glm::vec3(0, 0, -stepZ);
			}
		} else {
			if (tmax.y < tmax.z) {
				Y += stepY;
				if (Y == outY)
					return false;
				distance = tmax.y;
				tmax.y += tdelta.y;
				normal = glm::vec3(0, -stepY, 0);
			} else {
				Z += stepZ;
				if (Z == outZ)
					return false;
				distance = tmax.z;
				tmax.z += tdelta.z;
				normal = glm::vec3(0, 0, -stepZ);
			}
		}
	}
	return false;
}

__device__ inline bool intersect_voxel_simple(const ShadowQueue& ray, Scene::GPUScene scene, glm::uvec3 grid_dimensions) {
	//// Check if ray hits grid AABB
	//float tminn;
	//if (!intersect_aabb_branchless2(ray, tminn)) {
	//	//if (!intersect_aabb_correct(ray, tminn)) {
	//	return false;
	//}

	//// Move ray to hitpoint
	//glm::vec3 origin = ray.origin;
	//if (tminn > 0) {
	//	origin += ray.direction * tminn;

	//	constexpr glm::vec3 grid_center(grid_size / 2.f);
	//	glm::vec3 to_center = glm::abs(grid_center - origin);
	//	glm::vec3 signs = glm::sign(origin - grid_center);
	//	to_center /= glm::max(to_center.x, glm::max(to_center.y, to_center.z));
	//	glm::vec3 normal = signs * glm::trunc(to_center + 0.000001f);

	//	origin += -normal * epsilon;
	//}

	//// Initialize
	//glm::vec3 cb, tmax, tdelta;
	//int stepX, outX, X = ((int)origin.x); // / 8;
	//int stepY, outY, Y = ((int)origin.y); // / 8;
	//int stepZ, outZ, Z = ((int)origin.z); // / 8;
	////origin /= 8.f;

	//// Needed because sometimes the AABB intersect returns true while the ray is actually outside slightly. Only happens for faces that touch the AABB sides
	//if (X < 0 || X >= grid_size || Y < 0 || Y >= grid_size || Z < 0 || Z >= grid_size) {
	//	//printf("full X: %i Y: %i Z: %i Bounce:%i Innie:%i\n", X, Y, Z, ray.bounces, innie);
	//	return false;
	//}

	//if (ray.direction.x > 0) {
	//	stepX = 1;
	//	outX = grid_size;
	//	cb.x = (X + 1);
	//} else {
	//	stepX = -1;
	//	outX = -1;
	//	cb.x = X;
	//}
	//if (ray.direction.y > 0.0f) {
	//	stepY = 1;
	//	outY = grid_size;
	//	cb.y = (Y + 1);
	//} else {
	//	stepY = -1, outY = -1;
	//	cb.y = Y;
	//}
	//if (ray.direction.z > 0.0f) {
	//	stepZ = 1;
	//	outZ = grid_size;
	//	cb.z = (Z + 1);
	//} else {
	//	stepZ = -1;
	//	outZ = -1;
	//	cb.z = Z;
	//}
	//float rxr, ryr, rzr;
	//if (ray.direction.x != 0) {
	//	rxr = 1.0f / ray.direction.x;
	//	tmax.x = (cb.x - origin.x) * rxr;
	//	tdelta.x = stepX * rxr;
	//} else
	//	tmax.x = 1000000;
	//if (ray.direction.y != 0) {
	//	ryr = 1.0f / ray.direction.y;
	//	tmax.y = (cb.y - origin.y) * ryr;
	//	tdelta.y = stepY * ryr;
	//} else
	//	tmax.y = 1000000;
	//if (ray.direction.z != 0) {
	//	rzr = 1.0f / ray.direction.z;
	//	tmax.z = (cb.z - origin.z) * rzr;
	//	tdelta.z = stepZ * rzr;
	//} else
	//	tmax.z = 1000000;

	//// Stepping through grid
	//while (1) {
	//	//if (scene.voxels[X + Y * grid_size + Z * grid_size * grid_size]) {
	//	//	//printf("bounces:%i x:%f y:%f z:%f\n", ray.bounces, origin.x, origin.y, origin.z);
	//	//	ray.distance = distance + glm::max(tminn, 0.f) + epsilon;
	//	//	return true;
	//	//}

	//	if (scene.grid[X / 8 + (Y / 8) * cells + (Z / 8) * cells * cells] != nullptr) {
	//		return true;
	//	}

	//	if (tmax.x < tmax.y) {
	//		if (tmax.x < tmax.z) {
	//			X += stepX;
	//			if (X == outX)
	//				return false;
	//			tmax.x += tdelta.x;
	//		} else {
	//			Z += stepZ;
	//			if (Z == outZ)
	//				return false;
	//			tmax.z += tdelta.z;
	//		}
	//	} else {
	//		if (tmax.y < tmax.z) {
	//			Y += stepY;
	//			if (Y == outY)
	//				return false;
	//			tmax.y += tdelta.y;
	//		} else {
	//			Z += stepZ;
	//			if (Z == outZ)
	//				return false;
	//			tmax.z += tdelta.z;
	//		}
	//	}
	//}
	return false;
}

//__device__ inline bool intersect_scene(RayQueue& ray, Scene::GPUScene sceneData) {
//	float d;
//	ray.distance = VERY_FAR;
//
//	for (int i = NUM_SPHERES; i--;) {
//		//d = spheres[i].intersect(ray);
//		if ((d = spheres[i].intersect(ray)) && d < ray.distance) {
//			ray.distance = d;
//			ray.identifier = i;
//			//ray.geometry_type = GeometryType::Sphere;
//		}
//	}
//
//	glm::vec3 normal;
//	if (intersect_voxel(ray, sceneData, normal)) {
//		ray.identifier = 0;
//		//ray.geometry_type == GeometryType::Triangle;
//		ray.distance = 1;
//		return true;
//	}
//
//
//	//if (sceneData.CUDACachedBVH.intersect(ray)) {
//	//	ray.geometry_type = GeometryType::Triangle;
//	//}
//	return ray.distance < VERY_FAR;
//}

//__device__ inline bool intersect_scene_simple(ShadowQueue& ray, Scene::GPUScene sceneData, const float& closestAllowed) {
//	float d;
//
//	/*if (sceneData.CUDACachedBVH.intersectSimple(ray, closestAllowed))
//		return true;*/
//
//	for (int i = NUM_SPHERES; i--;) {
//		if ((d = spheres[i].intersect_simple(ray)) && (d + epsilon) < closestAllowed) {
//			return true;
//		}
//	}
//	return false;
//}

/*
	Given a direction unit vector W, computes two other vectors U and V which 
	make uvw an orthonormal basis.
*/
//TODO(Dan): Implement Frisvad method.
__forceinline __device__ void computeOrthonormalBasisNaive(const glm::vec3& w, glm::vec3* u, glm::vec3* v) {
	if (fabs(w.x) > .9) { /*If W is to close to X axis then pick Y*/
		*u = glm::vec3{ 0.0f, 1.0f, 0.0f };
	} else { /*Pick X axis*/
		*u = glm::vec3{ 1.0f, 0.0f, 0.0f };
	}
	*u = normalize(cross(*u, w));
	*v = cross(w, *u);
}
__device__ glm::vec2 ConcentricSampleDisk(const glm::vec2& u) {
	//Map from [0,1] to [-1,1]
	glm::vec2 uOffset = 2.f * u - glm::vec2(1, 1);

	// Handle degeneracy at the origin
	if (uOffset.x == 0 && uOffset.y == 0)
		return glm::vec2(0, 0);

	// Apply concentric mapping to point
	float theta, r;
	if (std::abs(uOffset.x) > std::abs(uOffset.y)) {
		r = uOffset.x;
		theta = pi / 4 * (uOffset.y / uOffset.x);
	} else {
		r = uOffset.y;
		theta = pi / 2 - pi / 4 * (uOffset.x / uOffset.y);
	}
	return r * glm::vec2(std::cosf(theta), std::sinf(theta));
}

//Number of rays still active after the shade kernel.
__device__ unsigned int primary_ray_cnt = 0;
//The index of the ray at which we start generating more rays in ray generation step.
//Effectively is the last index which was previously generated + 1.
__device__ unsigned int start_position = 0;
//Ray number incremented by every thread in primary_rays ray generation
__device__ unsigned int raynr_primary = 0;
//Ray number to fetch different ray from every CUDA thread during the extend step.
__device__ unsigned int raynr_extend = 0;
//Ray number to fetch different ray from every CUDA thread in the shade step.
__device__ unsigned int raynr_shade = 0;
//Ray number to fetch different ray from every CUDA thread in the connect step.
__device__ unsigned int raynr_connect = 0;
//Number of shadow rays generated in shade step, which are placed in connect step.
__device__ unsigned int shadow_ray_cnt = 0;

///Kernel should be called after primary ray generation but before other wavefront steps.
__global__ void set_wavefront_globals() {

	//Get how many rays we created last generation step.
	const unsigned int progress_last_frame = ray_queue_buffer_size - primary_ray_cnt;

	//The starting position for the next step is where we left off last time.
	//Last step we progressed from the start_position by progress_last_frame rays.
	//Next step we start from prev starting position incremented by how much we progressed this frame
	start_position += progress_last_frame;
	start_position = start_position % (render_width * render_height);
	//Zero out counters atomically incremented for all wavefront kernels.
	shadow_ray_cnt = 0;
	primary_ray_cnt = 0;
	raynr_primary = 0;
	raynr_extend = 0;
	raynr_shade = 0;
	raynr_connect = 0;
}

/// Generate primary rays. Fill ray_buffer up till max length.
__global__ void primary_rays(RayQueue* ray_buffer, glm::vec3 camera_right, glm::vec3 camera_up, glm::vec3 camera_direction, glm::vec3 O, unsigned int frame, float focalDistance, float lens_radius, Scene::GPUScene sceneData, glm::vec4* blit_buffer) {

	//Fill ray buffer up to ray_queue_buffer_size.
	while (true) {
		const unsigned int index = atomicAdd(&raynr_primary, 1);
		//Buffer already includes rays generated by previous "shade" step (primary_ray_cnt)
		const unsigned int ray_index_buffer = index + primary_ray_cnt;
		if (ray_index_buffer > ray_queue_buffer_size - 1) {
			return;
		}
		//Initialize random seed
		unsigned int seed = (frame * 147565741) * 720898027 * index;

		//Compute (x,y) coords based on position in buffer.
		// X goes (left -> right); Y goes (top -> bottom)

		const unsigned int x = (start_position + index) % render_width;
		const unsigned int y = ((start_position + index) / render_width) % render_height;

		//Get random stratified points inside pixel;
		glm::vec2 sample2D = Random2DStratifiedSample(seed);
		const float rand_point_pixelX = x - sample2D.x;
		const float rand_point_pixelY = y - sample2D.y;

#if 0 //Ordinary random points
		const float rand_point_pixelX = x - RandomFloat(seed);
		const float rand_point_pixelY = y - RandomFloat(seed);
#endif

		const float normalized_i = (rand_point_pixelX / (float)render_width) - 0.5f;
		const float normalized_j = ((render_height - rand_point_pixelY) / (float)render_height) - 0.5f;

		//Normal direction which we would compute even without DoF
		glm::vec3 directionToFocalPlane = camera_direction + normalized_i * camera_right + normalized_j * camera_up;
		directionToFocalPlane = glm::normalize(directionToFocalPlane);

		//Get the convergence point which is at focalDistance)
		//TODO(Dan): I currently multiply by 3 because I felt it would be easier for the ImGui slider.
		// Fix this by modifying how slider works?
		const int ImGui_slider_hack = 3.0f;
		glm::vec3 convergencePoint = O + focalDistance * ImGui_slider_hack * directionToFocalPlane;

		glm::vec2 lens_sample(RandomFloat(seed), RandomFloat(seed));
		glm::vec2 pLens = lens_radius * ConcentricSampleDisk(lens_sample);
		glm::vec3 newOrigin = O + camera_right * pLens.x + camera_up * pLens.y;

		glm::vec3 direction = glm::normalize(convergencePoint - newOrigin);

		ray_buffer[ray_index_buffer] = { newOrigin, direction, { 1, 1, 1 }, { 0, 0, 0 }, 0, 0, 0, y * render_width + x };
	}
}

/// Advance the ray segments once
__global__ void __launch_bounds__(128, 8) extend(RayQueue* ray_buffer, Scene::GPUScene sceneData, glm::vec4* blit_buffer, unsigned int seed) {
	while (true) {
		const unsigned int index = atomicAdd(&raynr_extend, 1);

		if (index > ray_queue_buffer_size - 1) {
			return;
		}
		RayQueue& ray = ray_buffer[index];

		ray.distance = VERY_FAR;
		//intersect_voxel(ray, sceneData, ray.normal, {32, 32, 32});

		if (intersect_voxel(ray.origin, ray.direction, ray.normal, ray.distance, sceneData)) {
			//glm::vec3 yoyo = ray.origin + ray.direction * ray.distance;
			//atomicAdd(&blit_buffer[ray.pixel_index].r, 1.f);
			//atomicAdd(&blit_buffer[ray.pixel_index].g, 1.f);
			//atomicAdd(&blit_buffer[ray.pixel_index].b, 1.f);
			atomicAdd(&blit_buffer[ray.pixel_index].r, ray.normal.x);
			atomicAdd(&blit_buffer[ray.pixel_index].g, ray.normal.y);
			atomicAdd(&blit_buffer[ray.pixel_index].b, ray.normal.z);
			atomicAdd(&blit_buffer[ray.pixel_index].a, 1.f);
			//printf("x:%f y:%f z:%f\n", ray.normal.x, ray.normal.y, ray.normal.z);
		}
	}
}

/// Process collisions and spawn extension and shadow rays.
/// Rays that continue get placed in ray_buffer_next to be processed next frame
__global__ void __launch_bounds__(128, 8) shade(RayQueue* ray_buffer, RayQueue* ray_buffer_next, ShadowQueue* shadowQueue, Scene::GPUScene sceneData, glm::vec4* blit_buffer, unsigned int frame) {

	while (true) {
		const unsigned int index = atomicAdd(&raynr_shade, 1);

		if (index > ray_queue_buffer_size - 1) {
			return;
		}

		int new_frame = 0;
		RayQueue& ray = ray_buffer[index];

		//Each iteration we add color to the blit_buffer.
		//Color can be non-zero if sun/sky or we're counting emisivity for different objects.
		glm::vec3 color = glm::vec3(0.f);
		glm::vec3 object_color;
		unsigned int seed = (frame * ray.pixel_index * 147565741) * 720898027 * index;
		int reflection_type = DIFF;

		if (ray.distance < VERY_FAR) {
			ray.origin += ray.direction * ray.distance;
			//Prevent self-intersection
			ray.origin += ray.normal * 2.f * epsilon;

			// Generate new shadow ray
			glm::vec3 sunSampleDir = getConeSample(sunDirection, 1.0f - sunAngularDiameterCos, seed);
			float sunLight = dot(ray.normal, sunSampleDir);

			// < 0.f means sun is behind the surface
			if (sunLight > 0.f) {
				unsigned shadow_index = atomicAdd(&shadow_ray_cnt, 1);
				shadowQueue[shadow_index] = { ray.origin, sunSampleDir, 2.0f * ray.direct * (sun(sunSampleDir) * sunLight * 1E-5f), ray.pixel_index };
			}

			if (ray.bounces < MAX_BOUNCES) {
#if 0 // Stratified sampling.
				glm::vec2 samples = Random2DStratifiedSample(seed); 
				float r1 = 2.f * pi * samples.x;
				float r2 = samples.y;
#else
				float r1 = 2.f * pi * RandomFloat(seed);
				float r2 = RandomFloat(seed);
#endif
				float r2s = sqrt(r2);

				// Transform to hemisphere coordinate system
				glm::vec3 u, v;
				computeOrthonormalBasisNaive(ray.normal, &u, &v);
				// Get sample on hemisphere
				ray.direction = glm::normalize(u * cos(r1) * r2s + v * sin(r1) * r2s + ray.normal * sqrt(1 - r2));
			}

			//Russian roullete
			float p = glm::min(1.0f, glm::max(ray.direct.z, glm::max(ray.direct.x, ray.direct.y)));
			if (ray.bounces < MAX_BOUNCES && p > (0 + epsilon) && RandomFloat(seed) <= p) {
				//Add rays into the next ray_buffer to be processed next frame
				ray.bounces++;
				ray.direct *= 1.0f / p;

				unsigned primary_index = atomicAdd(&primary_ray_cnt, 1);
				ray_buffer_next[primary_index] = ray;
			} else { // MAX BOUNCES
				new_frame++;
			}

		} else { //NOTHING HIT
			// Don't generate new extended ray. Directly add emmisivity of sun/sky.
			color += ray.direct * sunsky(ray.direction);
			new_frame++;
		}

		//Color is added every frame to buffer. However color can only be non-zero for sun/sky and if emmisive surface
		//was hit.
		//TODO(Dan): Perf increase if only add when != 0? How to interact with sky = black?
		atomicAdd(&blit_buffer[ray.pixel_index].r, color.r);
		atomicAdd(&blit_buffer[ray.pixel_index].g, color.g);
		atomicAdd(&blit_buffer[ray.pixel_index].b, color.b);
		atomicAdd(&blit_buffer[ray.pixel_index].a, new_frame);
	}
}

/// Proccess shadow rays
__global__ void __launch_bounds__(128, 8) connect(ShadowQueue* queue, Scene::GPUScene sceneData, glm::vec4* blit_buffer) {
	while (true) {
		const unsigned int index = atomicAdd(&raynr_connect, 1);

		if (index >= shadow_ray_cnt) {
			return;
		}

		ShadowQueue& ray = queue[index];

		if (!intersect_voxel_simple(ray, sceneData, { 32, 32, 32 })) {
			atomicAdd(&blit_buffer[ray.pixel_index].r, ray.color.r);
			atomicAdd(&blit_buffer[ray.pixel_index].g, ray.color.g);
			atomicAdd(&blit_buffer[ray.pixel_index].b, ray.color.b);
		}
	}
}

__global__ void blit_onto_framebuffer(glm::vec4* blit_buffer) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= render_width || y >= render_height) {
		return;
	}

	const int index = y * render_width + x;
	glm::vec4 color = blit_buffer[index];
	glm::vec4 cl = glm::vec4(color.r, color.g, color.b, 1) / color.a;
	cl.a = 1;

	surf2Dwrite<glm::vec4>(cl, surf, x * sizeof(glm::vec4), y, hipBoundaryModeZero);
	//surf2Dwrite<glm::vec4>(glm::pow(cl, glm::vec4(1.0f / 2.2f)), surf, x * sizeof(glm::vec4), y, hipBoundaryModeZero);
}

hipError_t launch_kernels(hipArray_const_t array, glm::vec4* blit_buffer, Scene::GPUScene sceneData, RayQueue* ray_buffer, RayQueue* ray_buffer_next, ShadowQueue* shadow_queue) {
	static bool first_time = true;
	static bool reset_buffer = false;
	static unsigned int frame = 1;

	if (first_time) {
		first_time = false;

		/*	Sphere sphere_data[NUM_SPHERES] = { { 16.5, { 0, 40, 16.5f }, { 1, 1, 1 }, { 0, 0, 0 }, DIFF },
											{ 16.5, { 40, 0, 16.5f }, { 0.5, 0.5, 0.06 }, { 0, 0, 0 }, DIFF },
											{ 16.5, { -40, -50, 36.5f }, { 0.6, 0.5, 0.4 }, { 0, 0, 0 }, DIFF },
											{ 16.5, { -40, -50, 16.5f }, { 0.6, 0.5, 0.4 }, { 0, 0, 0 }, DIFF },
											{ 1e4f, { 0, 0, -1e4f - 20 }, { 1, 1, 1 }, { 0, 0, 0 }, DIFF },
											{ 20, { 0, -80, 20 }, { 1.0, 0.0, 0.0 }, { 0, 0, 0 }, DIFF },
											{ 9, { 0, -80, 120.0f }, { 0.0, 1.0, 0.0 }, { 3, 3, 3 }, LIGHT } };
		hipMemcpyToSymbol(HIP_SYMBOL(spheres), sphere_data, NUM_SPHERES * sizeof(Sphere));*/

		float sun_angular = cos(sunSize * pi / 180.f);
		cuda(MemcpyToSymbol(sunAngularDiameterCos, &sun_angular, sizeof(float)));
	}

	hipError_t cuda_err;
	static glm::vec3 last_pos;
	static glm::vec3 last_dir;
	static float last_focaldistance = 1;
	static float last_lensradius = 0.02f;

	cuda_err = cuda(BindSurfaceToArray(surf, array));

	if (cuda_err) {
		return cuda_err;
	}

	const glm::vec3 camera_right = glm::normalize(glm::cross(camera.direction, camera.up)) * 1.5f * ((float)render_width / render_height);
	const glm::vec3 camera_up = glm::normalize(glm::cross(camera_right, camera.direction)) * 1.5f;

	reset_buffer = last_pos != camera.position || last_dir != camera.direction || last_focaldistance != camera.focalDistance || camera.lensRadius != last_lensradius;

	if (sun_position_changed) {
		sun_position_changed = false;
		reset_buffer = true;
		cuda(MemcpyToSymbol(SunPos, &sun_position, sizeof(glm::vec2)));
		glm::vec3 sun_direction = glm::normalize(fromSpherical((sun_position - glm::vec2(0.0, 0.5)) * glm::vec2(6.28f, 3.14f)));
		cuda(MemcpyToSymbol(sunDirection, &sun_direction, sizeof(glm::vec3)));
	}

	if (reset_buffer) {
		reset_buffer = false;
		hipMemset(blit_buffer, 0, render_width * render_height * sizeof(float4));

		int new_value = 0;
		cuda(MemcpyToSymbol(primary_ray_cnt, &new_value, sizeof(int)));
	}
	primary_rays<<<sm_cores * 8, 128>>>(ray_buffer, camera_right, camera_up, camera.direction, camera.position, frame, camera.focalDistance, camera.lensRadius, sceneData, blit_buffer);
	set_wavefront_globals<<<1, 1>>>();
	extend<<<sm_cores * 8, 128>>>(ray_buffer, sceneData, blit_buffer, frame);
	//shade<<<sm_cores * 8, 128>>>(ray_buffer, ray_buffer_next, shadow_queue, sceneData, blit_buffer, frame);
	//connect<<<sm_cores * 8, 128>>>(shadow_queue, sceneData, blit_buffer);

	dim3 threads = dim3(16, 16, 1);
	dim3 blocks = dim3(render_width / threads.x, render_height / threads.y, 1);
	blit_onto_framebuffer<<<blocks, threads>>>(blit_buffer);

	cuda(DeviceSynchronize());

	//Frame is used as XORSHIFT seed, but we must ensure it's not 0
	if (frame == UINT_MAX)
		frame = 0;

	frame++;

	//hold_frame++;
	last_pos = camera.position;
	last_dir = camera.direction;
	last_focaldistance = camera.focalDistance;
	last_lensradius = camera.lensRadius;

	return hipSuccess;
}